#include "hip/hip_runtime.h"
#include "miner/cuda/miner.h"

#include "miner/common/constants.h"

#include <cgbn/cgbn.h>
#include <gmpxx.h>

// NEEDS TO BE INCLUDED AFTER CGBN AND GMP
#include "cu_helpers.h"

#include <cassert>
#include <iostream>
#include <vector>

using namespace miner::common;
using namespace miner::cuda;

CudaMiner::CudaMiner(int device) : device_(device)
{
}

CudaMiner::~CudaMiner()
{
}

namespace kernel
{

static const uint32_t BIT_SIZE = 512;
static const uint32_t C_SIZE = 232;

typedef cgbn_mem_t<BIT_SIZE> bn_mem_t;

struct MimcParams
{
    bn_mem_t P;
    bn_mem_t *C;
    std::size_t C_size;
};

struct CudaWorkItem
{
    int64_t x;
    int64_t y;
    bool is_planet;
    bn_mem_t hash;
};

template <uint32_t tpi> struct BnParams
{
    static const uint32_t TPI = tpi; // GCBN threads per intstance.
};

void to_mpz(mpz_t r, const bn_mem_t &x)
{
    mpz_import(r, BIT_SIZE / 32, -1, sizeof(uint32_t), 0, 0, x._limbs);
}

void from_mpz(mpz_srcptr s, bn_mem_t &n)
{
    uint32_t count = BIT_SIZE / 32;
    uint32_t *x = n._limbs;
    size_t words;

    if (mpz_sizeinbase(s, 2) > count * 32)
    {
        fprintf(stderr, "from_mpz failed -- result does not fit\n");
        exit(1);
    }

    mpz_export(x, &words, -1, sizeof(uint32_t), 0, 0, s);
    while (words < count)
    {
        x[words++] = 0;
    }
}

template <class env_t, class bn_t = typename env_t::cgbn_t>
__device__ __forceinline__ void wrap_coordinate(env_t &env, bn_t &c_bn, int64_t c, const bn_t &p)
{
    typename env_t::cgbn_t n, m;
    if (c >= 0)
    {
        env.set_ui32(n, c);
        env.set(c_bn, n);
        return;
    }
    env.set_ui32(m, -c);
    env.sub(n, p, m);
    env.set(c_bn, n);
}

template <class env_t, class bn_t = typename env_t::cgbn_t>
__device__ __forceinline__ void field_add(env_t &env, bn_t &r, const bn_t &a, const bn_t &b, bn_t &t, const bn_t &p)
{
    env.add(t, a, b);
    env.rem(r, t, p);
}

template <class env_t, class bn_t = typename env_t::cgbn_t>
__device__ __forceinline__ void field_mul(env_t &env, bn_t &r, const bn_t &a, const bn_t &b, bn_t &t, const bn_t &p)
{
    env.mul(t, a, b);
    env.rem(r, t, p);
}

template <class env_t, class bn_t = typename env_t::cgbn_t>
__device__ __forceinline__ void fifth_power(env_t &env, bn_t &r, const bn_t &n, bn_t &s, bn_t &f, bn_t &t,
                                            const bn_t &p)
{
    field_mul(env, s, n, n, t, p);
    field_mul(env, f, s, s, t, p);
    field_mul(env, r, f, n, t, p);
}

template <class env_t, class bn_t = typename env_t::cgbn_t> class Sponge
{
  public:
    __device__ void reset(env_t &env)
    {
        env.set_ui32(l_, 0);
        env.set_ui32(r_, 0);
    }

    __device__ void inject(env_t &env, const bn_t &x, const bn_t &P)
    {
        field_add(env, l_, l_, x, t0_, P);
    }

    __device__ void mix(env_t &env, const bn_t &key, const bn_t *C, std::size_t C_size, const bn_t &P)
    {
        for (uint32_t j = 0; j < C_size; ++j)
        {
            field_add(env, t0_, key, l_, t1_, P);
            field_add(env, t1_, t0_, C[j], t2_, P);
            fifth_power(env, t0_, t1_, t2_, t3_, t4_, P);
            field_add(env, t1_, t0_, r_, t2_, P);
            env.set(r_, l_);
            env.set(l_, t1_);
        }
        field_add(env, t0_, key, l_, t2_, P);
        fifth_power(env, t1_, t0_, t2_, t3_, t4_, P);
        field_add(env, t0_, t1_, r_, t2_, P);
        env.set(r_, t0_);
    }

    __device__ void result(env_t &env, bn_t &out)
    {
        env.set(out, l_);
    }

  private:
    bn_t l_;
    bn_t r_;

    bn_t t0_;
    bn_t t1_;
    bn_t t2_;
    bn_t t3_;
    bn_t t4_;
};

template <class bn_params>
__global__ void mine_batch_kernel(cgbn_error_report_t *report, CudaWorkItem *batch, std::size_t batch_size,
                                  uint32_t items_per_thread, bn_mem_t planet_threshold_mem, bn_mem_t key_mem,
                                  bn_mem_t P_mem, bn_mem_t *C_mem, std::size_t C_size)
{
    using context_t = cgbn_context_t<bn_params::TPI>;
    using env_t = cgbn_env_t<context_t, BIT_SIZE>;
    using bn_t = typename env_t::cgbn_t;

    context_t ctx(cgbn_report_monitor, report);
    env_t env(ctx);

    uint32_t block_x = blockIdx.x;

    // Copy mimc constants to memory
    bn_t C[C_SIZE];
    for (uint32_t i = 0; i < C_size; ++i)
    {
        env.load(C[i], &(C_mem[i]));
    }

    bn_t P, key, planet_threshold;
    env.load(P, &P_mem);
    env.load(key, &key_mem);
    env.load(planet_threshold, &planet_threshold_mem);

    __syncthreads();

    Sponge<env_t> sponge;
    uint32_t idx;
    bn_t yi, xi, hash;
    for (std::size_t i = 0; i < items_per_thread; ++i)
    {
        idx = block_x * items_per_thread + i;
        if (idx < batch_size)
        {
            wrap_coordinate(env, xi, batch[idx].x, P);
            sponge.reset(env);
            sponge.inject(env, xi, P);
            sponge.mix(env, key, C, C_size, P);
            wrap_coordinate(env, yi, batch[idx].y, P);
            sponge.inject(env, yi, P);
            sponge.mix(env, key, C, C_size, P);
            sponge.result(env, hash);
            env.store(&(batch[idx].hash), hash);
            batch[idx].is_planet = env.compare(hash, planet_threshold) < 0;
        }
    }
}

template <class bn_params>
void run_mine_batch(int device, std::vector<WorkItem> &batch, const MimcParams &mimc, const bn_mem_t &planet_threshold,
                    const bn_mem_t &key)
{
    cgbn_error_report_t *bn_report;

    CUDA_CHECK(hipSetDevice(device));
    CUDA_CHECK(cgbn_error_report_alloc(&bn_report));

    bn_mem_t *d_C;
    CUDA_CHECK(hipMalloc(&d_C, sizeof(bn_mem_t) * mimc.C_size));
    CUDA_CHECK(hipMemcpy(d_C, mimc.C, sizeof(bn_mem_t) * mimc.C_size, hipMemcpyHostToDevice));

    CudaWorkItem *d_batch, *cpu_batch;
    cpu_batch = static_cast<CudaWorkItem *>(malloc(sizeof(CudaWorkItem) * batch.size()));
    CUDA_CHECK(hipMalloc(&d_batch, sizeof(CudaWorkItem) * batch.size()));

    for (std::size_t i = 0; i < batch.size(); ++i)
    {
        cpu_batch[i].x = batch[i].x;
        cpu_batch[i].y = batch[i].y;
        cpu_batch[i].is_planet = false;
    }

    CUDA_CHECK(hipMemcpy(d_batch, cpu_batch, sizeof(CudaWorkItem) * batch.size(), hipMemcpyHostToDevice));

    uint32_t items_per_thread = 128;

    // grid_size = ceil(size / items_per_thread)
    uint32_t grid_size = (batch.size() + items_per_thread - 1) / items_per_thread;
    dim3 block_size(bn_params::TPI);

    std::cout << "Starting kernel with grid size " << grid_size << std::endl;
    mine_batch_kernel<bn_params><<<grid_size, block_size>>>(bn_report, d_batch, batch.size(), items_per_thread,
                                                            planet_threshold, key, mimc.P, d_C, mimc.C_size);

    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipGetLastError());
    CGBN_CHECK(bn_report);

    CUDA_CHECK(hipMemcpy(cpu_batch, d_batch, sizeof(CudaWorkItem) * batch.size(), hipMemcpyDeviceToHost));

    mpz_class planet_hash;
    for (std::size_t i = 0; i < batch.size(); ++i)
    {
        assert(batch[i].x == cpu_batch[i].x && batch[i].y == cpu_batch[i].y);
        batch[i].is_planet = cpu_batch[i].is_planet;
        to_mpz(planet_hash.get_mpz_t(), cpu_batch[i].hash);
        batch[i].hash = planet_hash.get_str();
    }
}

} // namespace kernel

void CudaMiner::mine_batch(std::vector<common::WorkItem> &items, uint32_t rarity, uint32_t key) const
{
    kernel::bn_mem_t planet_threshold_bn, key_bn;

    mpz_class rarity_mpz(rarity);
    mpz_class planet_threshold = P / rarity_mpz;
    kernel::from_mpz(planet_threshold.get_mpz_t(), planet_threshold_bn);

    mpz_class key_mpz(key);
    kernel::from_mpz(key_mpz.get_mpz_t(), key_bn);

    kernel::bn_mem_t P_bn;
    kernel::bn_mem_t C_bn[kernel::C_SIZE];

    assert(C.size() <= kernel::C_SIZE);
    kernel::from_mpz(P.get_mpz_t(), P_bn);
    for (std::size_t i = 0; i < C.size(); i++)
    {
        kernel::from_mpz(C[i].get_mpz_t(), C_bn[i]);
    }

    kernel::MimcParams mimc{
        .P = P_bn,
        .C = C_bn,
        .C_size = C.size(),
    };

    typedef kernel::BnParams<32> bn_params_32;
    kernel::run_mine_batch<bn_params_32>(device_, items, mimc, planet_threshold_bn, key_bn);
}